
#include <hip/hip_runtime.h>

__global__ void complementDNA(int* d_x, int x) {
    x = x + 1;
    *d_x = x;
}

int _complementDNA(int x) {
    int* d_x;
    hipMalloc( (void**) &d_x, sizeof(int) );

    complementDNA<<<1,1>>>(d_x, x);

    int r;
    hipMemcpy(&r, d_x, sizeof(int), hipMemcpyDeviceToHost);

    return r;
}